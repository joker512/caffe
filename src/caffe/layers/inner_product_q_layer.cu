#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/filler.hpp"
#include "caffe/layers/inner_product_q_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
__global__ void set_kernel_2(const int num_output, const int quant_index, Dtype* d, int* b, Dtype* output) {
	CUDA_KERNEL_LOOP(index, num_output) {
		output[index] = d[b[quant_index * num_output + index]];
	}
}

template <typename Dtype>
void InnerProductQLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
                                            const vector<Blob<Dtype>*>& top) {
        const int K = this->layer_param_.inner_product_q_param().k();
        const int BITS = static_cast<int>(log2((float)K));
        const int TOTAL_BITS = 32;
        const int REST_BITS = TOTAL_BITS - BITS;
        const int M = this->layer_param_.inner_product_q_param().m();

        Dtype* bottom_data = bottom[0]->mutable_gpu_data();
        Dtype* top_data = top[0]->mutable_gpu_data();
        Dtype* D = this->blobs_[0]->mutable_gpu_data();
        const Dtype* bias = this->blobs_[1]->gpu_data();
        unsigned int* B_data = (unsigned int*)(this->blobs_[2]->cpu_data());
        int Bsize = num_output * num_input / M * sizeof(int);
        int* B = new int[Bsize];
        for (int i = 0, total_bit_shift = 0; i < num_output * num_input / M; ++i, total_bit_shift += BITS) {
            int byte_shift = total_bit_shift / TOTAL_BITS;
            int bit_shift = total_bit_shift % TOTAL_BITS;
            int shift = REST_BITS - bit_shift;
            B[i] = (shift < 0 ? B_data[byte_shift] << -shift | B_data[byte_shift + 1] >> (TOTAL_BITS + shift) :
                                B_data[byte_shift] >> shift) & (K - 1);
        }
        int* Bgpu = 0;
        hipMalloc((void**)&Bgpu, Bsize);
        hipMemcpy(Bgpu, B, Bsize, hipMemcpyHostToDevice);

        /*for (int i = 0; i < batch_size; ++i) {
            caffe_gpu_set<Dtype>(num_output, Dtype(0), top_data + i * num_output);
            for (int j = 0; j < num_input / M; ++j) {
                Dtype *S = bottom_data + i * num_input + j * M;
                Dtype *d = D + K * M * j;
                Dtype *output;
                Dtype *output2;
                hipMalloc((void**)&output, K * sizeof(Dtype));
                hipMalloc((void**)&output2, num_output * sizeof(Dtype));
                caffe_gpu_gemv<Dtype>(CblasTrans, M, K, (Dtype) 1., d, S, (Dtype) 0., output);
                set_kernel_2<Dtype><<<CAFFE_GET_BLOCKS(num_output), CAFFE_CUDA_NUM_THREADS>>>(num_output, j, output, Bgpu, output2);
                caffe_gpu_add<Dtype>(num_output, top_data + i * num_output, output2, top_data + i * num_output);
                hipFree(output);
                hipFree(output2);
            }
        }*/
        hipFree(Bgpu);
        delete[] B;
        caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, batch_size, num_output, 1, (Dtype)1.,
                              bias_multiplier_.gpu_data(), bias, (Dtype)1., top_data);
}

template <typename Dtype>
void InnerProductQLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
}

INSTANTIATE_LAYER_GPU_FUNCS(InnerProductQLayer);

}
