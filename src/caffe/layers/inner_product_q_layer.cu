#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/filler.hpp"
#include "caffe/layers/inner_product_q_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
void InnerProductQLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
        const int K = this->layer_param_.inner_product_q_param().k();
        const int BITS = static_cast<int>(log2((float)K));
        const int TOTAL_BITS = 8;
        const int REST_BITS = TOTAL_BITS - BITS;
        const int M = this->layer_param_.inner_product_q_param().m();

        Dtype* bottom_data = bottom[0]->mutable_cpu_data();
        Dtype* top_data = top[0]->mutable_cpu_data();
        Dtype* D = this->blobs_[0]->mutable_cpu_data();
        const Dtype* bias = this->blobs_[1]->cpu_data();
        unsigned char* B_data = (unsigned char*)(this->blobs_[2]->cpu_data());
        int* B = new int[num_output * num_input / M];
        for (int i = 0, total_bit_shift = 0; i < num_output * num_input / M; ++i, total_bit_shift += BITS) {
            int byte_shift = total_bit_shift / TOTAL_BITS;
            int bit_shift = total_bit_shift % TOTAL_BITS;
            int shift = REST_BITS - bit_shift;
            B[i] = static_cast<int>((shift < 0 ? B_data[byte_shift] << -shift | B_data[byte_shift + 1] >> (TOTAL_BITS + shift) :
                                              B_data[byte_shift] >> shift) & (K - 1));
        }
        for (int i = 0; i < batch_size; ++i) {
            caffe_set(num_output, Dtype(0), top_data + i * num_output);
            for (int j = 0; j < num_input / M; ++j) {
                Dtype *S = bottom_data + i * num_input + j * M;
                Dtype *d = D + K * M * j;
                Dtype *output = new Dtype[K];
                caffe_gpu_gemv(CblasTrans, K, M, (Dtype) 1., d, S, (Dtype) 1., output);
		LOG(ERROR) << d[0] << " " << d[1] << " " << d[2] << " " << d[3];
		LOG(ERROR) << S[0] << " " << S[1];
		LOG(ERROR) << output[0] << " " << output[1];
                for (int l = 0; l < num_output; ++l) {
                    top_data[i * num_output + l] += output[B[j * num_output + l]];
                }
                delete[] output;
            }
        }
        delete[] B;
        //caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, num_input, num_output, 1, (Dtype)1.,
        //                      bias_multiplier_.cpu_data(), bias, (Dtype)1., top_data);
}

template <typename Dtype>
void InnerProductQLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
}

INSTANTIATE_LAYER_GPU_FUNCS(InnerProductQLayer);

}
